
#include <hip/hip_runtime.h>
__global__
void trilinear_interpolation(int n,int axis,int * dim,float *x, float *y, float *z,float *intepolated_flow,float *vol)
{
    // x,y,z  1d size = (h*w*d) array positions where you want to find flow values
    //n total number of voxels in vol h*w*d
    //vol flow field carrying displacement in one direction  1d size = (h*w*d)
    //intepolated_flow storing output  1d size = (h*w*d)

  int i = blockIdx.x*blockDim.x + threadIdx.x;
  
  if (i < n) {
    int H=dim[0];
    int W=dim[1];
    int D=dim[2];
    float max =dim[axis];
    float next_pos=0.0;
    float X=x[i];
    float Y=y[i];
    float Z=z[i];
    if(axis==0){
      next_pos=X;
    }if(axis==1){
      next_pos=Y;
    }if(axis==2){
      next_pos=Z;
    }
    int X0=floor(X);
    int X1=ceil(X);
    int Y0=floor(Y);
    int Y1=ceil(Y);
    int Z0=floor(Z);
    int Z1=ceil(Z);
    float d_x=0;
    float d_y=0;
    float d_z=0;
    if (X0<0){
      //printf("%d \n", X0);
      X0=0;
    }
    if (X1<0){
      X1=0;
      ////printf("%d \n", X1);

    }
    if (X0>H-1){
      X0=H-1;
      //printf("%d \n", X0);
    }
    if (X1>H-1){
      X1=H-1;
      //printf("%d \n", X1);
    }
    if (Y0<0){
      Y0=0;
      //printf("%d \n", Y0);
    }
    if (Y1<0){
      Y1=0;
    //printf("%d \n", Y1);
    }
    if (Y0>W-1){
      Y0=W-1;
      //printf("%d \n", Y0);
    }
    if (Y1>W-1){
      Y1=W-1;
      //printf("%d \n", Y1);
    }
    if (Z0<0){
      Z0=0;
      //printf("%d \n", Z0);
    }
    if (Z1<0){
      Z1=0;
      //printf("%d \n", Z1);
    }
    if (Z0>D-1){
      Z0=D-1;
      //printf("%d \n", Z0);

    }
    if (Z1>D-1){
      Z1=D-1;
      //printf("%d \n", Z1);
    }

    if(X0!=X1){ 
        d_x=(X-X0)/(X1-X);
    }
    if(Y0!=Y1){
        d_y=(Y-Y0)/(Y1-Y);
    }
    if(Z0!=Z1){
        d_z=(Z-Z0)/(Z1-Z);
    }
    float c00,c01,c10,c11,c0,c1,c;
    c00=vol[X0*W*D+Y0*D+Z0]*(1-d_x)+vol[X1*W*D+Y0*D+Z0]*d_x;
    c01=vol[X0*W*D+Y0*D+Z1]*(1-d_x)+vol[X1*W*D+Y0*D+Z1]*d_x;
    c10=vol[X0*W*D+Y1*D+Z0]*(1-d_x)+vol[X1*W*D+Y1*D+Z0]*d_x;
    c11=vol[X0*W*D+Y1*D+Z1]*(1-d_x)+vol[X1*W*D+Y1*D+Z1]*d_x;


    c0=c00*(1-d_y)+c10*d_y;
    c1=c01*(1-d_y)+c11*d_y;

    c=c0*(1-d_z)+c1*d_z;
    c=c+next_pos;
    if(c >= max-1){
      c=0.0f;
    }
    if(c <= 0.0f){
      c=0.0f;
    }

    intepolated_flow[i]=c;

  }

}